#include "hip/hip_runtime.h"
#include "graph.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define DEBUG(x)
#define N_THREADS_PER_BLOCK (1 << 5)

Graph *initGraph(int numVertices, GraphDensity density)
{
  Graph *graph = (Graph *)malloc(sizeof(Graph));
  graph->numVertices = numVertices;
  graph->numEdges = 0; // This will be calculated in generateEdges

  // Allocate memory for arrays
  graph->adjacencyList = (int *)malloc(numVertices * numVertices * sizeof(int));
  graph->edgesOffset = (int *)malloc(numVertices * sizeof(int));
  graph->edgesSize = (int *)malloc(numVertices * sizeof(int));

  // Initialize arrays
  for (int i = 0; i < numVertices; i++)
  {
    graph->edgesOffset[i] = i * numVertices;
    graph->edgesSize[i] = 0;
  }

  // Generate edges based on density
  generateEdges(graph, density);

  return graph;
}
void generateEdges(Graph *graph, GraphDensity density)
{
  srand(time(NULL)); // Seed the random number generator
  int maxAdditionalEdges;

  // First, create a chain to ensure connectivity
  for (int i = 0; i < graph->numVertices - 1; i++)
  {
    // Connect vertex i to i + 1
    graph->adjacencyList[graph->edgesOffset[i] + graph->edgesSize[i]] = i + 1;
    graph->edgesSize[i]++;
    graph->numEdges++;
  }

  // Determine the maximum number of additional edges based on density
  switch (density)
  {
  case Sparse:
    maxAdditionalEdges = graph->numVertices;
    break;
  case Medium:
    maxAdditionalEdges = graph->numVertices * 2;
    break;
  case Dense:
    maxAdditionalEdges = graph->numVertices * (graph->numVertices - 1) / 2;
    break;
  }

  // Add additional edges
  for (int i = 0; i < maxAdditionalEdges; i++)
  {
    int v = rand() % graph->numVertices;
    int w = rand() % graph->numVertices;

    // Avoid self-loops and duplicate edges
    if (v != w && graph->edgesSize[v] < graph->numVertices - 1)
    {
      // Add edge from v to w
      graph->adjacencyList[graph->edgesOffset[v] + graph->edgesSize[v]] = w;
      graph->edgesSize[v]++;

      // Increment the total edge count
      graph->numEdges++;
    }
  }
}

// Function to free the graph from memory
void freeGraph(Graph *graph)
{
  free(graph->adjacencyList);
  free(graph->edgesOffset);
  free(graph->edgesSize);
  free(graph);
}

__global__ void initializeDeviceArray(int n, int *d_arr, int value, int start_index)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == start_index)
  {
    d_arr[start_index] = 0;
  }
  else if (tid < n)
  {
    d_arr[tid] = value;
  }
}

__global__ void printDeviceArray(int *d_arr, int n)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n)
  {
    printf("d_arr[%i] = %i \n", tid, d_arr[tid]);
  }
}

__global__ void computeNextQueue(int *adjacencyList, int *edgesOffset, int *edgesSize, int *distance,
                                 int queueSize, int *currentQueue, int *nextQueueSize, int *nextQueue, int level)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < queueSize)
  {
    int current = currentQueue[tid];
    for (int i = edgesOffset[current]; i < edgesOffset[current] + edgesSize[current]; ++i)
    {
      int v = adjacencyList[i];
      if (distance[v] == INT_MAX)
      {
        distance[v] = level + 1;
        int position = atomicAdd(nextQueueSize, 1);
        nextQueue[position] = v;
      }
    }
  }
}
void bfsGPU(int start, Graph *G, int *distance)
{
  const int n_blocks = (G->numVertices + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK;

  // Initialization of GPU variables
  int *d_adjacencyList, *d_edgesOffset, *d_edgesSize, *d_firstQueue, *d_secondQueue, *d_nextQueueSize, *d_distance;

  // Allocation on device
  const int size = G->numVertices * sizeof(int);
  const int adjacencySize = G->numVertices * G->numVertices * sizeof(int); // Assuming the adjacency list is a full matrix
  hipMalloc((void **)&d_adjacencyList, adjacencySize);
  hipMalloc((void **)&d_edgesOffset, size);
  hipMalloc((void **)&d_edgesSize, size);
  hipMalloc((void **)&d_firstQueue, size);
  hipMalloc((void **)&d_secondQueue, size);
  hipMalloc((void **)&d_distance, size);
  hipMalloc((void **)&d_nextQueueSize, sizeof(int));

  // Copy inputs to device
  hipMemcpy(d_adjacencyList, G->adjacencyList, adjacencySize, hipMemcpyHostToDevice);
  hipMemcpy(d_edgesOffset, G->edgesOffset, size, hipMemcpyHostToDevice);
  hipMemcpy(d_edgesSize, G->edgesSize, size, hipMemcpyHostToDevice);

  // Initialize distance array on host and copy to device
  for (int i = 0; i < G->numVertices; ++i)
  {
    distance[i] = INT_MAX;
  }
  distance[start] = 0;
  hipMemcpy(d_distance, distance, size, hipMemcpyHostToDevice);

  int currentQueueSize = 1, level = 0;

  while (currentQueueSize > 0)
  {
    int *d_currentQueue, *d_nextQueue;
    if (level % 2 == 0)
    {
      d_currentQueue = d_firstQueue;
      d_nextQueue = d_secondQueue;
    }
    else
    {
      d_currentQueue = d_secondQueue;
      d_nextQueue = d_firstQueue;
    }

    computeNextQueue<<<n_blocks, N_THREADS_PER_BLOCK>>>(d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance,
                                                        currentQueueSize, d_currentQueue, d_nextQueueSize, d_nextQueue, level);
    hipDeviceSynchronize();
    ++level;
    hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost);
    int resetQueueSize = 0;
    hipMemcpy(d_nextQueueSize, &resetQueueSize, sizeof(int), hipMemcpyHostToDevice);
  }

  hipMemcpy(distance, d_distance, size, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_adjacencyList);
  hipFree(d_edgesOffset);
  hipFree(d_edgesSize);
  hipFree(d_firstQueue);
  hipFree(d_secondQueue);
  hipFree(d_distance);
  hipFree(d_nextQueueSize);
}
int main()
{
  int numVertices = 500; // Example number of vertices
  Graph *myGraph = initGraph(numVertices, Sparse);

  // printf("Graph's Adjacency List:\n");
  // printGraph(myGraph);

  // Allocate memory for BFS distance and visited arrays
  int *distance = (int *)malloc(numVertices * sizeof(int));
  int *visited = (int *)malloc(numVertices * sizeof(int));
  clock_t startSerial, endSerial, startParallel, endParallel;
  startParallel = clock();
  bfsGPU(0, myGraph, distance);
  endParallel = clock();
  double timeTakenParallel = (double)(endParallel - startParallel) / CLOCKS_PER_SEC;
  printf("Cuda BFS took %f seconds.\n", timeTakenParallel);
}
